#include <iostream>
#include <fstream>
#include <chrono>
#include <cmath>
#include <hip/hip_runtime.h>

using namespace std;
using namespace std::chrono;

__global__ void solveSKernel(double x, double epsilon, double *result, int *converged, int maxIterations)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= maxIterations || *converged <= idx)
        return; // Прерываем выполнение, если достигнута сходимость

    double base = (x - 1) / (x + 1);
    double term = 1.0 / (2 * idx + 1) * pow(base, 2 * idx + 1);

    // Атомарное сложение для накопления результата
    atomicAdd(result, term);

    // Проверяем условие сходимости
    if (fabs(term) < epsilon)
    {
        atomicMin(converged, idx + 1); // Обновляем минимальное количество итераций
    }
}

double solveS(double x, double epsilon, int &iterations)
{
    const int maxIterations = 100000; // Ограничение на количество итераций
    double result = 0.0;
    int converged = maxIterations;

    // Выделяем память на устройстве
    double *d_result;
    int *d_converged;
    hipMalloc((void **)&d_result, sizeof(double));
    hipMalloc((void **)&d_converged, sizeof(int));

    // Инициализируем значения на устройстве
    hipMemset(d_result, 0, sizeof(double));
    hipMemcpy(d_converged, &converged, sizeof(int), hipMemcpyHostToDevice);

    // Конфигурация CUDA
    int threadsPerBlock = 256;
    int blocksPerGrid = (maxIterations + threadsPerBlock - 1) / threadsPerBlock;

    // Запуск CUDA kernel
    solveSKernel<<<blocksPerGrid, threadsPerBlock>>>(x, epsilon, d_result, d_converged, maxIterations);

    // Синхронизация устройства
    hipDeviceSynchronize();

    // Копируем результат обратно на хост
    hipMemcpy(&result, d_result, sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(&converged, d_converged, sizeof(int), hipMemcpyDeviceToHost);

    // Освобождаем память на устройстве
    hipFree(d_result);
    hipFree(d_converged);

    // Устанавливаем количество итераций
    iterations = converged;
    return result;
}

// Вычисление Y(x) (не требует распараллеливания)
double solveY(double x)
{
    return log(x) / 2;
}

int main()
{
    double a, b, h, epsilon;

    cout << "Введите a: ";
    cin >> a;
    cout << "Введите b: ";
    cin >> b;
    cout << "Введите шаг h: ";
    cin >> h;
    cout << "Введите epsilon: ";
    cin >> epsilon;

    ofstream outputFile("results.txt");
    if (!outputFile)
    {
        cerr << "Ошибка открытия файла!" << endl;
        return 1;
    }

    cout << " x | Y(x) | S(x) | Итерации | Время (сек) " << endl;
    cout << "-------------------------------------------------" << endl;

    for (double x = a; x <= b; x += h)
    {
        if (x <= 0)
            continue;
        if (h <= 0 && epsilon <= 0)
            break;

        int iterations = 0;
        double resultY = solveY(x);

        auto start = high_resolution_clock::now();
        double resultS = solveS(x, epsilon, iterations);
        auto end = high_resolution_clock::now();

        duration<double> elapsedTime = end - start;

        cout << x << " | " << resultY << " | " << resultS << " | " << iterations << " | " << elapsedTime.count() << endl;
        outputFile << iterations << " | " << elapsedTime.count() << endl;
    }

    outputFile.close();
    return 0;
}